#include "CompressedSparseRow.h"

#include "Algebra/Modules/Vectors/HostArrays.h"
#include "HostCompressedSparseRow.h"
#include "Utilities/HandleUnsuccessfulCuSparseCall.h"
#include "Utilities/HandleUnsuccessfulCudaCall.h"

#include <cstddef> // std::size_t
#include <hip/hip_runtime.h> // hipFree, hipMalloc
#include <hipsparse.h> // cuSparseCreateCsr
#include <iostream> // std::cerr

using Algebra::Modules::Vectors::HostArray;
using Utilities::HandleUnsuccessfulCUDACall;
using Utilities::HandleUnsuccessfulCuSparseCall;
using std::cerr;
using std::size_t;

namespace Algebra
{
namespace Modules
{
namespace Matrices
{
namespace SparseMatrices
{

CompressedSparseRowMatrix::CompressedSparseRowMatrix(
  const size_t M,
  const size_t N,
  const size_t number_of_elements
  ):
  d_values_{nullptr},
  d_columns_{nullptr},
  d_rows_{nullptr},
  M_{M},
  N_{N},
  number_of_elements_{number_of_elements},
  matrix_handler_{nullptr}  
{
  const hipError_t err_values {
    hipMalloc(
      reinterpret_cast<void**>(&d_values_),
      number_of_elements_ * sizeof(float))};
  const hipError_t err_columns {
    hipMalloc(
      reinterpret_cast<void**>(&d_columns_),
      number_of_elements_ * sizeof(int))};
  const hipError_t err_rows {
    hipMalloc(
      reinterpret_cast<void**>(&d_rows_),
      (M + 1) * sizeof(int))};

  if (err_values != hipSuccess)
  {
    cerr << "Failed to allocate device array for values (error code " <<
      hipGetErrorString(err_values) << ")!\n";
  }  

  if (err_columns != hipSuccess)
  {
    cerr << "Failed to allocate device array for column indices (error code " <<
      hipGetErrorString(err_columns) << ")!\n";
  }  

  if (err_rows != hipSuccess)
  {
    cerr << "Failed to allocate device array for row indices (error code " <<
      hipGetErrorString(err_rows) << ")!\n";
  }  

  // ref. https://docs.nvidia.com/cuda/cusparse/index.html#cusparsecreatecsr
  // hipsparseCreateCsr initializes sparse matrix descriptor hipsparseSpMatDescr_t
  // spMatDescr in CSR format.
  const hipsparseStatus_t create_sparse_status {hipsparseCreateCsr(
    &matrix_handler_,
    // number of rows
    M,
    // number of columns
    N,
    // Number of non-zero entries of sparse matrix.
    number_of_elements_,
    // Row offsets of sparse matrix.
    d_rows_,
    d_columns_,
    d_values_,
    // Data type of csrRowOffsets.
    HIPSPARSE_INDEX_32I,
    // Data type of csrColInd.
    HIPSPARSE_INDEX_32I,
    // Index base of csrRowOffsets and csrColInd.
    HIPSPARSE_INDEX_BASE_ZERO,
    // Datatype of csrValues.
    HIP_R_32F)};

  if (create_sparse_status != HIPSPARSE_STATUS_SUCCESS)
  {
    cerr << "Failed to create Sparse CSR (error code " <<
      // ref. https://docs.nvidia.com/cuda/cusparse/index.html#cusparsegeterrorstring
      // const char* cusparseGetErrorSTring(hipsparseStatus_t status).
      hipsparseGetErrorString(create_sparse_status) << ")!\n";    
  }
}

CompressedSparseRowMatrix::~CompressedSparseRowMatrix()
{
  const hipError_t err_values {hipFree(d_values_)};
  const hipError_t err_columns {hipFree(d_columns_)};
  const hipError_t err_rows {hipFree(d_rows_)};

  if (err_values != hipSuccess)
  {
    cerr << "Failed to free device array for values (error code " <<
      hipGetErrorString(err_values) << ")!\n";
  }
  if (err_columns != hipSuccess)
  {
    cerr << "Failed to free device array for column indicies (error code " <<
      hipGetErrorString(err_columns) << ")!\n";
  }
  if (err_rows != hipSuccess)
  {
    cerr << "Failed to free device array for row indices (error code " <<
      hipGetErrorString(err_rows) << ")!\n";
  }

  if (matrix_handler_)
  {
    // ref. https://docs.nvidia.com/cuda/cusparse/index.html#cusparsedestroyspmat
    // Releases host memory allocated for sparse matrix descriptor spMatDescr.
    const hipsparseStatus_t destroy_matrix_status {hipsparseDestroySpMat(
      matrix_handler_)};

    if (destroy_matrix_status != HIPSPARSE_STATUS_SUCCESS)
    {
      cerr << "Failed to destroy Sparse CSR descriptor (error code " <<
        hipsparseGetErrorString(destroy_matrix_status) << ")!\n";    
    }
  }

  // We choose not to throw upon a failed garbage clean up.
}

void CompressedSparseRowMatrix::copy_host_input_to_device(
  const HostCompressedSparseRowMatrix& h_a)
{
  HandleUnsuccessfulCUDACall handle_columns {
    "Failed to copy column indices from host to device"};

  handle_columns(hipMemcpy(
    d_columns_,
    h_a.J_,
    h_a.number_of_elements_ * sizeof(int),
    hipMemcpyHostToDevice));

  HandleUnsuccessfulCUDACall handle_rows {
    "Failed to copy row from host to device"};

  handle_rows(hipMemcpy(
    d_rows_,
    h_a.I_,
    (h_a.M_ + 1) * sizeof(int),
    hipMemcpyHostToDevice));

  HandleUnsuccessfulCUDACall handle_values {
    "Failed to copy values from host to device"};

  handle_values(hipMemcpy(
    d_values_,
    h_a.values_,
    h_a.number_of_elements_ * sizeof(float),
    hipMemcpyHostToDevice));
}

void CompressedSparseRowMatrix::copy_device_output_to_host(
  HostCompressedSparseRowMatrix& h_a)
{
  HandleUnsuccessfulCUDACall handle_columns {
    "Failed to copy column indices from device to host"};

  handle_columns(hipMemcpy(
    h_a.J_,
    d_columns_,
    number_of_elements_ * sizeof(int),
    hipMemcpyDeviceToHost));

  HandleUnsuccessfulCUDACall handle_rows {
    "Failed to copy row from device to host"};

  handle_rows(hipMemcpy(
    h_a.I_,
    d_rows_,
    (M_ + 1) * sizeof(int),
    hipMemcpyDeviceToHost));

  HandleUnsuccessfulCUDACall handle_values {
    "Failed to copy values from device to host"};

  handle_values(hipMemcpy(
    h_a.values_,
    d_values_,
    number_of_elements_ * sizeof(float),
    hipMemcpyDeviceToHost));  
}

DenseVector::DenseVector(
  const size_t N
  ):
  d_values_{nullptr},
  number_of_elements_{N},
  vector_descriptor_{nullptr}  
{
  HandleUnsuccessfulCUDACall handle_cuda_malloc {
    "Failed to allocate device array for values"};

  handle_cuda_malloc(
    hipMalloc(
      reinterpret_cast<void**>(&d_values_),
      number_of_elements_ * sizeof(float)));

  // ref. https://docs.nvidia.com/cuda/cusparse/index.html#cusparsecreatednvec
  // Initializes dense vector descriptor.
  const hipsparseStatus_t create_dense_vector_status {
    hipsparseCreateDnVec(
      &vector_descriptor_,
      number_of_elements_,
      d_values_,
      HIP_R_32F)};

  if (create_dense_vector_status != HIPSPARSE_STATUS_SUCCESS)
  {
    cerr << "Failed to create dense vector (error code " <<
      hipsparseGetErrorString(create_dense_vector_status) << ")!\n";
  }
}

DenseVector::~DenseVector()
{
  const hipError_t err_values {hipFree(d_values_)};

  HandleUnsuccessfulCUDACall handle_cuda_free {
    "Failed to free device array for values"};

  handle_cuda_free(err_values);

  if (vector_descriptor_)
  {
    HandleUnsuccessfulCuSparseCall handle_destroy_dense_vector {
      "Failed to destroy dense vector"};
    
    handle_destroy_dense_vector(hipsparseDestroyDnVec(vector_descriptor_));
  }
}

void DenseVector::copy_host_input_to_device(const HostArray& h_a)
{
  HandleUnsuccessfulCUDACall handle_values {
    "Failed to copy values from host to device"};

  handle_values(hipMemcpy(
    d_values_,
    h_a.values_,
    h_a.number_of_elements_ * sizeof(float),
    hipMemcpyHostToDevice));
}

void DenseVector::copy_device_output_to_host(HostArray& h_a)
{
  HandleUnsuccessfulCUDACall handle_values {
    "Failed to copy values from device to host"};

  handle_values(hipMemcpy(
    h_a.values_,
    d_values_,
    number_of_elements_ * sizeof(float),
    hipMemcpyDeviceToHost));  
}

} // namespace SparseMatrices
} // namespace Matrices
} // namespace Modules
} // namespace Algebra
