#include "DeviceVectorAdditionArrays.h"

#include <cstddef> // std::size_t
#include <cstdlib> // free
#include <iostream> // std::cerr
#include <stdexcept>

using std::cerr;
using std::size_t;

namespace Algebra
{
namespace Modules
{
namespace Vectors
{

DeviceVectorAdditionArrays::DeviceVectorAdditionArrays(
  const std::size_t input_size
  ):
  number_of_elements_{input_size},
  d_A_{nullptr},
  d_B_{nullptr},
  d_C_{nullptr}
{
  const size_t size_in_bytes {input_size * sizeof(float)};

  const hipError_t err_A {
    hipMalloc(reinterpret_cast<void**>(&d_A_), size_in_bytes)};
  const hipError_t err_B {
    hipMalloc(reinterpret_cast<void**>(&d_B_), size_in_bytes)};
  const hipError_t err_C {
    hipMalloc(reinterpret_cast<void**>(&d_C_), size_in_bytes)};

  if (err_A != hipSuccess)
  {
    cerr << "Failed to allocate device array A (error code " <<
      hipGetErrorString(err_A) << ")!\n";
  }
  if (err_B != hipSuccess)
  {
    cerr << "Failed to allocate device array B (error code " <<
      hipGetErrorString(err_B) << ")!\n";
  }
  if (err_C != hipSuccess)
  {
    cerr << "Failed to allocate device array C (error code " <<
      hipGetErrorString(err_C) << ")!\n";
  }

  if (err_A != hipSuccess || err_B != hipSuccess || err_C != hipSuccess)
  {
    throw std::runtime_error("Failed to allocate device array");
  }
}

DeviceVectorAdditionArrays::~DeviceVectorAdditionArrays()
{
  const hipError_t err_A {hipFree(d_A_)};
  const hipError_t err_B {hipFree(d_B_)};
  const hipError_t err_C {hipFree(d_C_)};

  if (err_A != hipSuccess)
  {
    cerr << "Failed to free device array A (error code " <<
      hipGetErrorString(err_A) << ")!\n";
  }
  if (err_B != hipSuccess)
  {
    cerr << "Failed to free device array B (error code " <<
      hipGetErrorString(err_B) << ")!\n";
  }
  if (err_C != hipSuccess)
  {
    cerr << "Failed to free device array C (error code " <<
      hipGetErrorString(err_C) << ")!\n";
  }

  // We choose not to throw upon a failed garbage clean up.
}

} // namespace Vectors
} // namespace Modules
} // namespace Algebra