#include "hip/hip_runtime.h"
#include "DeviceVectorAdditionArrays.h"
#include "HostVectorAdditionArrays.h"
#include "VectorAddition.h"

#include <cstddef> // std::size_t
#include <cstdint>
#include <iostream> // std::cerr
#include <hip/hip_runtime.h>

using std::cerr;
using std::size_t;

namespace Algebra
{
namespace Modules
{
namespace Vectors
{

__global__ void vector_addition(
  const float* A,
  const float* B,
  float* C,
  size_t number_of_elements)
{
	const std::size_t i {blockDim.x * blockIdx.x + threadIdx.x};

  if (i < number_of_elements)
  {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

void vector_addition(
  HostVectorAdditionArrays& h_arrays,
  DeviceVectorAdditionArrays& d_arrays,
  const std::size_t threads_per_block)
{
  copy_host_input_to_device(h_arrays, d_arrays);

  const size_t blocks_per_grid {
    (h_arrays.number_of_elements_ + threads_per_block - 1) / threads_per_block};

  vector_addition<<<blocks_per_grid, threads_per_block>>>(
    d_arrays.d_A_,
    d_arrays.d_B_,
    d_arrays.d_C_,
    d_arrays.number_of_elements_);

  const hipError_t err {hipGetLastError()};

  if (err != hipSuccess)
  {
    cerr << "Failed to launch vector_addition kernel (error code " <<
      hipGetErrorString(err) << ")!\n";
  }

  copy_device_output_to_host(d_arrays, h_arrays);
}

void copy_host_input_to_device(
  const HostVectorAdditionArrays& hab,
  DeviceVectorAdditionArrays& dab)
{
  const hipError_t err_A {
    hipMemcpy(
      dab.d_A_,
      hab.h_A_,
      hab.number_of_elements_ * sizeof(float),
      hipMemcpyHostToDevice)};

  const hipError_t err_B {
    hipMemcpy(
      dab.d_B_,
      hab.h_B_,
      hab.number_of_elements_ * sizeof(float),
      hipMemcpyHostToDevice)};

  if (err_A != hipSuccess)
  {
    cerr << "Failed to copy array A from host to device (error code " <<
      hipGetErrorString(err_A) << ")!\n";
  }
  if (err_B != hipSuccess)
  {
    cerr << "Failed to copy array B from host to device (error code " <<
      hipGetErrorString(err_B) << ")!\n";
  }
}

void copy_device_output_to_host(
  const DeviceVectorAdditionArrays& dc,
  HostVectorAdditionArrays& hc)
{
  const hipError_t err_C {
    hipMemcpy(
      hc.h_C_,
      dc.d_C_,
      dc.number_of_elements_ * sizeof(float),
      hipMemcpyDeviceToHost)};

  if (err_C != hipSuccess)
  {
    cerr << "Failed to copy array C from device to host (error code " <<
      hipGetErrorString(err_C) << ")!\n";
  }
}

} // namespace Vectors
} // namespace Modules
} // namespace Algebra
