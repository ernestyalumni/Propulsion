#include "SparseMatrixMorphism.h"

#include "Algebra/Modules/Matrices/CompressedSparseRow.h"
#include "Utilities/HandleUnsuccessfulCuSparseCall.h"
#include "Utilities/HandleUnsuccessfulCudaCall.h"

#include <hipsparse.h> // hipsparseSpMatDescr_t

using Algebra::Modules::Matrices::SparseMatrices::CompressedSparseRowMatrix;
using Utilities::HandleUnsuccessfulCUDACall;
using Utilities::HandleUnsuccessfulCuSparseCall;

namespace Algebra
{
namespace Modules
{
namespace Morphisms
{

SparseMatrixMorphismOnDenseVectors::SparseMatrixMorphismOnDenseVectors(
  const float alpha,
  const float beta
  ):
  buffer_{nullptr},
  cusparse_handle_{0},
  buffer_size_{0},
  alpha_{alpha},
  beta_{beta}
{
  HandleUnsuccessfulCuSparseCall handle_create_handle {
    "Failed to create cuSparse handle"};

  // https://docs.nvidia.com/cuda/cusparse/index.html#cusparsecreate
  // hipsparseStatus_t hipsparseCreate(hipsparseHandle_t* handle) initializes
  // cuSparse library and creates handle on cuSparse context. It allocates
  // hardware resources necessary for accessing GPU.
  handle_create_handle(hipsparseCreate(&cusparse_handle_));
}

SparseMatrixMorphismOnDenseVectors::~SparseMatrixMorphismOnDenseVectors()
{
  HandleUnsuccessfulCuSparseCall handle_destroy_handle {
    "Failed to destroy cuSparse handle"};

  // ref: https://docs.nvidia.com/cuda/cusparse/index.html#cusparsedestroy
  // Releases CPU-side resources used by cuSparse library. Release of GPU-side
  // resources maybe deferred until application shuts down.
  handle_destroy_handle(hipsparseDestroy(cusparse_handle_));    

  HandleUnsuccessfulCUDACall handle_free_buffer {"Failed to free buffer"};

  handle_free_buffer(hipFree(buffer_));
}

bool SparseMatrixMorphismOnDenseVectors::linear_transform(
  CompressedSparseRowMatrix& A,
  DenseVector& x,
  DenseVector& y)
{
  HandleUnsuccessfulCuSparseCall handle_multiplication {
    "Failed to multiply with sparse matrix and dense vector"};

  // Performs multiplication of a sparse matrix and dense vector.

  handle_multiplication(hipsparseSpMV(
    cusparse_handle_,
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    &alpha_,
    A.matrix_descriptor_,
    x.vector_descriptor_,
    &beta_,
    y.vector_descriptor_,
    HIP_R_32F,
    HIPSPARSE_SPMV_ALG_DEFAULT,
    buffer_));

  return handle_multiplication.is_cusparse_success();
}

bool SparseMatrixMorphismOnDenseVectors::buffer_size(
  CompressedSparseRowMatrix& A,
  DenseVector& x,
  DenseVector& y)
{
  HandleUnsuccessfulCuSparseCall handle_buffer_size {"Failed to buffer size"};

  // See https://docs.nvidia.com/cuda/cusparse/index.html#cusparsespmv
  handle_buffer_size(hipsparseSpMV_bufferSize(
    cusparse_handle_,
    // op(A) = A for HIPSPARSE_OPERATION_NON_TRANSPOSE.
    HIPSPARSE_OPERATION_NON_TRANSPOSE,
    &alpha_,
    A.matrix_descriptor_,
    x.vector_descriptor_,
    &beta_,
    y.vector_descriptor_,
    // hipDataType computeType; mixed regular, complex computation.
    HIP_R_32F,
    // hipsparseSpMVAlg_t alg - algorithm for computation.
    HIPSPARSE_SPMV_ALG_DEFAULT,
    // void* externalBuffer - pointer to a workspace buffer of at least
    // bufferSize bytes.
    &buffer_size_));

  HandleUnsuccessfulCUDACall handle_allocate_buffer {
    "Failed to allocate for buffer"};

  handle_allocate_buffer(hipMalloc(&buffer_, buffer_size_));

  return handle_buffer_size.is_cusparse_success() &&
    handle_allocate_buffer.is_cuda_success();
}

} // namespace Morphisms
} // namespace Modules
} // namespace Algebra
