#include "hip/hip_runtime.h"
#include "DataStructures/Array.h"
#include "gtest/gtest.h"

#include <algorithm>
#include <cmath>
#include <cstddef>
#include <vector>

using DataStructures::Array;
using std::size_t;
using std::vector;

namespace GoogleUnitTests
{
namespace DataStructures
{

// We'll use the example code from
// https://forums.developer.nvidia.com/t/using-glfw-library-in-a-cuda-program/248434
// "Using GLFW library in a CUDA program"

__global__ void fill_RGB(unsigned char* rgb)
{
  const size_t index {blockIdx.x * blockDim.x + threadIdx.x};
  const size_t offset {index * 3};
  // Red value
  rgb[offset] = index % 255;
  // Green value
  rgb[offset + 1] = (index * 3) % 255;
  // Blue value
  rgb[offset + 2] = (index * 7) % 255;
}

constexpr size_t example_width {800};
constexpr size_t example_height {600};

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ArrayTests, Constructible)
{
  Array<unsigned char> array {example_width * example_height * 3};

  SUCCEED();
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ArrayTests, CopiesFromHostToDevice)
{
  const size_t N {8};

  Array<float> array {N};

  vector<float> source {};
  source.reserve(N);
  std::generate_n(
    std::back_inserter(source),
    N,
    [exponent = 0]() mutable
    {
      return std::pow(2.f, exponent++);
    });

  for (size_t i {0}; i < N; ++i)
  {
    EXPECT_EQ(source.at(i), std::pow(2.f, i));    
  }

  ASSERT_EQ(source.size(), N);

  EXPECT_TRUE(array.copy_host_input_to_device(source));

  vector<float> result (N);
  std::fill(result.begin(), result.end(), 1.f);
  array.copy_device_output_to_host(result);

  for (size_t i {0}; i < N; ++i)
  {
    EXPECT_EQ(result.at(i), std::pow(2.f, i));
  }
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ArrayTests, CUDAKernelFunctionCanMutateArray)
{
  Array<unsigned char> array {example_width * example_height * 3};
  const size_t threads_per_block {256};
  const size_t blocks_per_grid {
    (example_width * example_height + threads_per_block - 1) /
      threads_per_block};

  fill_RGB<<<blocks_per_grid, threads_per_block>>>(array.elements_);

  vector<unsigned char> host_vec_rgb (
    example_width * example_height * 3);

  EXPECT_TRUE(array.copy_device_output_to_host(host_vec_rgb));

  EXPECT_EQ(
    host_vec_rgb.size(),
    example_width * example_height * 3);

  for (size_t i {0}; i < threads_per_block; ++i)
  {
    for (size_t j {0}; j < blocks_per_grid; ++j)
    {
      const size_t index {i + j * threads_per_block};
      const size_t offset {index * 3};

      EXPECT_EQ(host_vec_rgb.at(offset), index % 255);
      EXPECT_EQ(host_vec_rgb.at(offset + 1), (index * 3) % 255);
      EXPECT_EQ(host_vec_rgb.at(offset + 2), (index * 7) % 255);
    }
  }
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ArrayTests, CanCopyToHostArray)
{
  Array<unsigned char> array {800 * 600 * 3};
  const size_t threads_per_block {256};
  const size_t blocks_per_grid {
    (800 * 600 + threads_per_block - 1) / threads_per_block};

  fill_RGB<<<blocks_per_grid, threads_per_block>>>(array.elements_);

  unsigned char* host_array = new unsigned char[
    example_width *
      example_height * 3];

  EXPECT_TRUE(array.copy_device_output_to_host(host_array));

  for (size_t i {0}; i < threads_per_block; ++i)
  {
    for (size_t j {0}; j < blocks_per_grid; ++j)
    {
      const size_t index {i + j * threads_per_block};
      const size_t offset {index * 3};

      EXPECT_EQ(host_array[offset], index % 255);
      EXPECT_EQ(host_array[offset + 1], (index * 3) % 255);
      EXPECT_EQ(host_array[offset + 2], (index * 7) % 255);
    }
  }

  delete[] host_array;
}

} // namespace DataStructures
} // namespace GoogleUnitTests