#include "hip/hip_runtime.h"
#include "Stencil.h"
#include <hip/hip_fp16.h> // For __half

namespace Testing
{
namespace Manifolds
{
namespace Operators
{
namespace FiniteDifference
{
namespace TwoDimensional
{

template struct Stencil<float, 1>;
template struct Stencil<float, 2>;
template struct Stencil<float, 3>;
template struct Stencil<float, 4>;

template struct Stencil<double, 1>;
template struct Stencil<double, 2>;
template struct Stencil<double, 3>;
template struct Stencil<double, 4>;

template struct Stencil<__half, 1>;
template struct Stencil<__half, 2>;
template struct Stencil<__half, 3>;
template struct Stencil<__half, 4>;

} // namespace TwoDimensional
} // namespace FiniteDifference
} // namespace Operators
} // namespace Manifolds
} // namespace Testing