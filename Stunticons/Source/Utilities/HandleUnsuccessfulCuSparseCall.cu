#include "HandleUnsuccessfulCuSparseCall.h"

#include <hipsparse.h>
#include <iostream> // std::cerr
#include <string>

using std::cerr;

namespace Utilities
{

HandleUnsuccessfulCuSparseCall::HandleUnsuccessfulCuSparseCall(
  const std::string& error_message
  ):
  error_message_{error_message},
  cusparse_status_{HIPSPARSE_STATUS_SUCCESS}
{}

void HandleUnsuccessfulCuSparseCall::operator()(
  const hipsparseStatus_t cusparse_status)
{
  cusparse_status_ = cusparse_status;

  if (!is_cusparse_success())
  {
    cerr << error_message_ << " (error code " <<
      // ref. https://docs.nvidia.com/cuda/cusparse/index.html#cusparsegeterrorstring
      // const char* cusparseGetErrorSTring(hipsparseStatus_t status).
      hipsparseGetErrorString(cusparse_status_) << ")!\n";
  }
}

} // namespace Utilities