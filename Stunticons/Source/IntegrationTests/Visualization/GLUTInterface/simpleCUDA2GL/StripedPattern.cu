#include "StripedPattern.h"
#include "Visualization/GLUTInterface/GLUTWindow.h"
#include "Visualization/OpenGLInterface/HandleGLError.h"

#include <hip/hip_runtime.h>
#include <GL/glut.h> // GLUT_RGBA

using GLUTWindowParameters =
  Visualization::GLUTInterface::GLUTWindow::Parameters;
using Visualization::OpenGLInterface::HandleGLError;

namespace IntegrationTests
{
namespace Visualization
{
namespace GLUTInterface
{

StripedPattern::Parameters::Parameters(
  const unsigned int image_width,
  const unsigned int image_height,
  const dim3 blocks
  ):
  image_width_{image_width},
  image_height_{image_height},
  blocks_{blocks},
  threads_{image_width / blocks.x, image_height / blocks.y, 1}
{}

const StripedPattern::Parameters StripedPattern::default_parameters_{
  512,
  512,
  dim3{16, 16, 1}};

const GLUTWindowParameters
  StripedPattern::default_glut_window_parameters_{
    "CUDA OpenGL post-processing",
    512,
    512,
    {GLUT_RGBA, GLUT_ALPHA, GLUT_DOUBLE, GLUT_DEPTH}};

void StripedPattern::run(int* argcp, char** argv)
{
  // Start of "initGL"

  ::Visualization::GLUTInterface::GLUTWindow::instance().initialize_glut(
    argcp,
    argv,
    default_glut_window_parameters_);

  ::Visualization::GLUTInterface::GLUTWindow::clear_color_buffers(
    0.5,
    0.5,
    0.5,
    1.0);

  // viewport
  glViewport(0, 0, 512, 512);

  // projection
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  gluPerspective(60.0, (GLfloat)512 / (GLfloat)512, 0.1f,
                 10.0f);

  glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

  glEnable(GL_LIGHT0);
  float red[] = {1.0f, 0.1f, 0.1f, 1.0f};
  float white[] = {1.0f, 1.0f, 1.0f, 1.0f};
  glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, red);
  glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
  glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 60.0f);

  HandleGLError gl_err {};

  // End of "initGL"

}

} // namespace GLUTInterface
} // namespace Visualization
} // namespace IntegrationTests