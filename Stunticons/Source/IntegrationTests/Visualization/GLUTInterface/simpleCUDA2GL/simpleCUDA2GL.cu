#include "hip/hip_runtime.h"
#include "simpleCUDA2GL.h"

#include <cstddef>

using std::size_t;

namespace IntegrationTests
{
namespace Visualization
{
namespace GLUTInterface
{

//------------------------------------------------------------------------------
/// \brief Clamp or bound a value x between a and b.
//------------------------------------------------------------------------------

__device__ float clamp(const float x, const float a, const float b)
{
  return max(a, min(b, x));
}

__device__ int clamp(const int x, const int a, const int b)
{
  return max(a, min(b, x));
}

//------------------------------------------------------------------------------
/// \brief Convert floating point RGB color to 8-bit integer.
//------------------------------------------------------------------------------
__device__ int rgb_to_int(float r, float g, float b)
{
  static constexpr float lower_bound {0.0f};
  static constexpr float upper_bound {255.0f};

  r = clamp(r, lower_bound, upper_bound);
  g = clamp(g, lower_bound, upper_bound);
  b = clamp(b, lower_bound, upper_bound);

  return (
    static_cast<int>(b) << 16 |
    (static_cast<int>(g) << 8) |
    static_cast<int>(r));
}

__global__ void make_striped_pattern(unsigned int* data, const int image_width)
{
  const size_t tx {threadIdx.x};
  const size_t ty {threadIdx.y};
  const size_t bw {blockIdx.x};
  const size_t bh {blockIdx.y};
  const size_t x {blockIdx.x * bw + tx};
  const size_t y {blockIdx.y * bh + ty};

  uchar4 c4 {make_uchar4((x & 0x20) ? 100 : 0, 0, (y & 0x20) ? 100 : 0, 0)};

  data[y * image_width + x] = rgb_to_int(c4.z, c4.y, c4.x);
}

void make_striped_pattern(
  const dim3 threads,
  const dim3 blocks,
  unsigned int* data,
  const int image_width)
{
  make_striped_pattern<<<threads, blocks>>>(data, image_width);
}

} // namespace GLUTInterface
} // namespace Visualization
} // namespace IntegrationTests