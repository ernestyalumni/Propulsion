#include "hip/hip_runtime.h"
#include "JuliaSet.h"

#include "IntegrationTests/Visualization/GLUTInterface/JuliaSet/IsInJuliaSet.h"
#include "IntegrationTests/Visualization/GLUTInterface/JuliaSet/Parameters.h"
#include "Visualization/CUDAGraphicsResource.h"
#include "Visualization/GLUTInterface/GLUTWindow.h"
#include "Visualization/MappedDevicePointer.h"
#include "Visualization/OpenGLInterface/BufferObjectNames.h"
#include "Visualization/OpenGLInterface/CreateOpenGLBuffer.h"
#include "Visualization/OpenGLInterface/DrawPixels.h"
#include "Visualization/OpenGLInterface/HandleGLError.h"

#include <GL/glut.h> // GLUT_RGBA
#include <hip/hip_runtime.h>
#include <functional>

using IntegrationTests::Visualization::GLUTInterface::JuliaSet::
  get_default_julia_parameters;

using GLUTWindowParameters =
  Visualization::GLUTInterface::GLUTWindow::Parameters;
using BufferObjectParameters =
  Visualization::OpenGLInterface::BufferObjectNames::Parameters;
using Visualization::CUDAGraphicsResource;
using Visualization::MappedDevicePointer;
using Visualization::OpenGLInterface::BufferObjectNames;
using Visualization::OpenGLInterface::CreateOpenGLBuffer;
using Visualization::OpenGLInterface::DrawPixels;
using Visualization::OpenGLInterface::HandleGLError;

namespace IntegrationTests
{
namespace Visualization
{
namespace GLUTInterface
{
namespace JuliaSet
{

JuliaSet::Parameters::Parameters(
  const unsigned int image_width,
  const unsigned int image_height,
  const dim3 blocks
  ):
  image_width_{image_width},
  image_height_{image_height},
  blocks_{blocks},
  threads_{image_width / blocks.x, image_height / blocks.y, 1}
{}

const JuliaSet::Parameters JuliaSet::default_parameters_{
  dimensions_,
  dimensions_,
  1};

const GLUTWindowParameters
  JuliaSet::default_glut_window_parameters_{
    "Julia Sets bit map",
    dimensions_,
    dimensions_,
    {GLUT_DOUBLE, GLUT_RGBA}};

void JuliaSet::draw_function()
{
  DrawPixels::draw_pixels_to_frame_buffer(
    DrawPixels::Parameters {
      JuliaSet::dimensions_,
      JuliaSet::dimensions_,
      GL_RGBA,
      GL_UNSIGNED_BYTE
    });

  DrawPixels::swap_buffers();
}

bool JuliaSet::run(int* argcp, char** argv)
{
  bool no_error {true};

  float scale {1.5};

  // Start of "initGL"

  ::Visualization::GLUTInterface::GLUTWindow::instance().initialize_glut(
    argcp,
    argv,
    default_glut_window_parameters_);

  HandleGLError gl_err {};

  // End of "initGL"

  BufferObjectParameters buffer_parameters {};
  buffer_parameters.binding_target_ = GL_PIXEL_UNPACK_BUFFER_ARB;
  buffer_parameters.usage_ = GL_DYNAMIC_DRAW_ARB;
  buffer_parameters.width_ = dimensions_;
  buffer_parameters.height_ = dimensions_;

  BufferObjectNames buffer_object {buffer_parameters};
  buffer_object.initialize();
  
  CreateOpenGLBuffer create_buffer {};
  no_error &= create_buffer.create_buffer_object_data(buffer_parameters);

  CUDAGraphicsResource cuda_graphics_resource {};
  const CUDAGraphicsResource::Parameters cuda_parameters {};
  cuda_graphics_resource.register_buffer_object(
    cuda_parameters,
    buffer_object);

  cuda_graphics_resource.map_resource();

  MappedDevicePointer<uchar4> mapped_device_pointer {};
  mapped_device_pointer.get_mapped_device_pointer(cuda_graphics_resource);

  is_in_julia_set<<<parameters_.threads_, parameters_.blocks_>>>(
    mapped_device_pointer.device_pointer_,
    scale,
    get_default_julia_parameters(dimensions_, dimensions_));

  return no_error;
}

void JuliaSet::display_and_exit(CUDAGraphicsResource& cuda_graphics_resource)
{
  cuda_graphics_resource.unmap_resource();

  glutDisplayFunc(draw_function);
  glutMainLoop();
}

} // namespace JuliaSet
} // namespace GLUTInterface
} // namespace Visualization
} // namespace IntegrationTests