#include "Utilities/HandleUnsuccessfulCudaCall.h"

// hipGraphicsGLRegisterBuffer, cudaGraphicsMapFlagNone
#include <cuda_gl_interop.h> 
#include <hip/hip_runtime.h> // hipFree, hipMalloc, hipMemcpyAsync
#include <GL/gl.h> // GLuint

int main()
{
  GLuint buffer_object {};
  hipGraphicsResource* resource {nullptr};

  hipGraphicsGLRegisterBuffer(
    &resource,
    buffer_object,
    cudaGraphicsMapFlagsNone);	
}